/*!
 * Copyright (c) 2017 by Contributors
 * \file quantized_conv2d.cu
 * \brief
 * \author Ziheng Jiang
*/
#include "./quantized_conv2d-inl.h"
#include "./quantization_utils.h"
#include "../tensor/matrix_op-inl.h"

namespace mxnet {
namespace op {

template<typename SrcType, typename DstType, typename CmpType>
class QuantizedConv2DCuDNNOp : public Operator {
 public:
  explicit QuantizedConv2DCuDNNOp(const Context& ctx,
                                  const std::vector<TShape>& in_shape,
                                  const std::vector<TShape>& out_shape,
                                  const QuantizedConv2DParam& param) {
    param_ = param;
    if (param_.layout == mshadow::kNCHW) {
      N = 0, H = 2, W = 3, C = 1;
    } else if (param_.layout == mshadow::kNHWC) {
      N = 0, H = 1, W = 2, C = 3;
    }
    src_type_ = mshadow::DataType<SrcType>::kCudnnFlag;
    dst_type_ = mshadow::DataType<DstType>::kCudnnFlag;
    cmp_type_ = mshadow::DataType<CmpType>::kCudnnFlag;
    algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    format_ = HIPDNN_TENSOR_NHWC;
    init_temp_size_ = false;
    InitDescriptors(ctx, in_shape, out_shape);
  }

  ~QuantizedConv2DCuDNNOp() {
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc_));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(data_desc_));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc_));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc_));
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_args) {
    using namespace mshadow;
    CHECK_EQ(in_data.size(), 6U);
    CHECK_EQ(out_data.size(), 3U);
    Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(s->dnn_handle_ownership_, Stream<gpu>::OwnHandle);

    TBlob data   = in_data[0];
    TBlob filter = in_data[1];
    TBlob out    = out_data[0];
    const TShape& dshape = data.shape_;
    const TShape& fshape = filter.shape_;
    const TShape& oshape = out.shape_;

    int res_cnt = 0;
    // allocate workspace
    if (!init_temp_size_) GetTempSize(ctx);
    Tensor<gpu, 1, SrcType> workspace =
      ctx.requested[res_cnt++].get_space_typed<gpu, 1, SrcType>(mshadow::Shape1(workspace_), s);

    if (param_.layout == mshadow::kNCHW) {
      TBlob data_(ctx.requested[res_cnt++].get_space_typed<gpu, 4, SrcType>(
          mshadow::Shape4(dshape[N], dshape[H], dshape[W], dshape[C]), s));
      TBlob filter_(ctx.requested[res_cnt++].get_space_typed<gpu, 4, SrcType>(
          mshadow::Shape4(fshape[N], fshape[H], fshape[W], fshape[C]), s));

      // input:  [NCHW] => [NHWC](batch, in_height, in_width, in_channels)
      // filter: [NCHW] => [NHWC](out_channels, filter_height, filter_width, in_channels)
      TransposeImpl<gpu>(ctx.run_ctx, data,   data_,   TShape({N, H, W, C}));
      TransposeImpl<gpu>(ctx.run_ctx, filter, filter_, TShape({N, H, W, C}));

      TBlob out_(ctx.requested[res_cnt++].get_space_typed<gpu, 4, DstType>(
          mshadow::Shape4(oshape[N], oshape[H], oshape[W], oshape[C]), s));
      TBlob out_tcast(ctx.requested[res_cnt++].get_space_typed<gpu, 4, int32_t>(
          mshadow::Shape4(oshape[N], oshape[H], oshape[W], oshape[C]), s));
      // input:  [NHWC](batch, in_height, in_width, in_channels)
      // filter: [HWNC](out_channels, filter_height, filter_width, in_channels)
      // output: [NHWC](batch, out_height, out_width, out_channels)

      CUDNN_CALL(hipdnnConvolutionForward(s->dnn_handle_,
                                         &alpha_,
                                         data_desc_,
                                         data_.dptr_,
                                         filter_desc_,
                                         filter_.dptr_,
                                         conv_desc_,
                                         algo_,
                                         workspace.dptr_,
                                         workspace_byte_,
                                         &beta_,
                                         out_desc_,
                                         out_.dptr_));

      Tensor<gpu, 1, DstType> out_tensor = out_.FlatTo1D<gpu, DstType>(s);
      Tensor<gpu, 1, int32_t> out_tcast_tensor = out_tcast.FlatTo1D<gpu, int32_t>(s);
      Assign(out_tcast_tensor, kWriteTo, mshadow::expr::tcast<int32_t>(out_tensor));
      // output: [NHWC](batch, out_height, out_width, out_channels) => [NCHW]
      TransposeImpl<gpu>(ctx.run_ctx, out_tcast, out, TShape({0, 3, 1, 2}));
    } else if (param_.layout == mshadow::kNHWC) {
      TBlob out_float(ctx.requested[res_cnt++].get_space_typed<gpu, 4, DstType>(
          mshadow::Shape4(oshape[N], oshape[H], oshape[W], oshape[C]), s));
      CUDNN_CALL(hipdnnConvolutionForward(s->dnn_handle_,
                                         &alpha_,
                                         data_desc_,
                                         data.dptr_,
                                         filter_desc_,
                                         filter.dptr_,
                                         conv_desc_,
                                         algo_,
                                         workspace.dptr_,
                                         workspace_byte_,
                                         &beta_,
                                         out_desc_,
                                         out_float.dptr_));
      Tensor<gpu, 1, DstType> out_float_tensor = out_float.FlatTo1D<gpu, DstType>(s);
      Tensor<gpu, 1, int32_t> out_tensor = out.FlatTo1D<gpu, int32_t>(s);
      Assign(out_tensor, kWriteTo, mshadow::expr::tcast<int32_t>(out_float_tensor));
    }

    mxnet_op::Kernel<QuantizationRangeForMultiplicationStruct, gpu>::Launch(s, 1,
      out_data[1].dptr<float>(), out_data[2].dptr<float>(),
       in_data[2].dptr<float>(),  in_data[3].dptr<float>(),
       in_data[4].dptr<float>(),  in_data[5].dptr<float>());
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_args) {
    LOG(FATAL) << "Not implemented";
  }


  void InitDescriptors(const Context& ctx,
                       const std::vector<TShape>& in_shape,
                       const std::vector<TShape>& out_shape) {
    TShape dshape =  in_shape[0];
    TShape kshape =  in_shape[1];
    TShape oshape = out_shape[0];
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&data_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc_));
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc_));

    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc_,
                                               param_.pad[0],
                                               param_.pad[1],
                                               param_.stride[0],
                                               param_.stride[1],
                                               1,
                                               1,
                                               HIPDNN_CROSS_CORRELATION,
                                               cmp_type_));

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(data_desc_,
                                          format_,
                                          src_type_,
                                          dshape[N],
                                          dshape[C],
                                          dshape[H],
                                          dshape[W]));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc_,
                                          format_,
                                          dst_type_,
                                          oshape[N],
                                          oshape[C],
                                          oshape[H],
                                          oshape[W]));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc_,
                                          src_type_,
                                          format_,
                                          kshape[N],
                                          kshape[C],
                                          kshape[H],
                                          kshape[W]));
  }

  void GetTempSize(const OpContext& ctx) {
    CHECK(!init_temp_size_)
      << "GetTempSize should only be called once.";
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(s->dnn_handle_,
                                                       data_desc_,
                                                       filter_desc_,
                                                       conv_desc_,
                                                       out_desc_,
                                                       algo_,
                                                       &workspace_byte_));
    workspace_ = workspace_byte_ / sizeof(SrcType) + 1;
    init_temp_size_ = true;
  }


 private:
  bool init_temp_size_ = false;
  QuantizedConv2DParam param_;
  size_t workspace_;
  size_t workspace_byte_;
  hipdnnDataType_t src_type_;
  hipdnnDataType_t dst_type_;
  hipdnnDataType_t cmp_type_;
  hipdnnTensorFormat_t format_;
  hipdnnConvolutionDescriptor_t conv_desc_;
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnFilterDescriptor_t filter_desc_;
  hipdnnTensorDescriptor_t out_desc_;
  hipdnnConvolutionFwdAlgo_t algo_;
  uint32_t N, H, W, C;
  float alpha_ = 1.0f;
  float beta_ = 0.0f;
};  // class QuantizedReluCuDNNOp


template<>
Operator* CreateOp<gpu>(int dtype,
                        const Context& ctx,
                        const std::vector<TShape>& in_shape,
                        const std::vector<TShape>& out_shape,
                        const QuantizedConv2DParam& param) {
  Operator *op = NULL;
  op = new QuantizedConv2DCuDNNOp<int8_t, float, int32_t>(ctx,
    in_shape, out_shape, param);
  return op;
}

}  // namespace op
}  // namespace mxnet


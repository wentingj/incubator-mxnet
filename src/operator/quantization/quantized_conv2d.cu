#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2017 by Contributors
 * \file quantized_conv2d.cu
 * \brief
 * \author Ziheng Jiang
*/
#include "./quantized_conv2d-inl.h"
#include "./quantization_utils.h"
#include "../tensor/matrix_op-inl.h"

namespace mxnet {
namespace op {

// value + bias_value * (range1 / limit_range1) * (limit_range2 / range2)
struct QuantizedBiasAddStruct {
  MSHADOW_XINLINE static void Map(int i, size_t bias_size, int32_t *out,
    const int8_t *bias, const float *min_out, const float *max_out,
    const float *min_bias, const float *max_bias, const size_t spatial_size) {
    float float_for_one_out_quant  =
      MaxAbs(*min_out, *max_out) / static_cast<double>(MaxValue<int32_t>());
    float float_for_one_bias_quant =
      MaxAbs(*min_bias, *max_bias) / static_cast<double>(MaxValue<int8_t>());
    const size_t channel_id = (i / spatial_size) % bias_size;
    out[i] = (out[i] * float_for_one_out_quant +
              bias[channel_id] * float_for_one_bias_quant) /
             float_for_one_out_quant;
  }
};

template<typename SrcType, typename DstType, typename CmpType>
class QuantizedConv2DCuDNNOp : public Operator {
 public:
  explicit QuantizedConv2DCuDNNOp(const Context& ctx,
                                  const std::vector<TShape>& in_shape,
                                  const std::vector<TShape>& out_shape,
                                  const QuantizedConv2DParam& param) {
    param_ = param;
    if (param_.layout == mshadow::kNCHW) {
      N = 0, H = 2, W = 3, C = 1;
    } else if (param_.layout == mshadow::kNHWC) {
      N = 0, H = 1, W = 2, C = 3;
    }
    src_type_ = mshadow::DataType<SrcType>::kCudnnFlag;
    dst_type_ = mshadow::DataType<DstType>::kCudnnFlag;
    cmp_type_ = mshadow::DataType<CmpType>::kCudnnFlag;
    algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    format_ = HIPDNN_TENSOR_NHWC;
    init_temp_size_ = false;
    InitDescriptors(ctx, in_shape, out_shape);
  }

  ~QuantizedConv2DCuDNNOp() {
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc_));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(data_desc_));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc_));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc_));
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_args) {
    using namespace mshadow;
    CHECK_EQ(in_data.size(), param_.no_bias? 6U : 9U);
    CHECK_EQ(out_data.size(), 3U);
    Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(s->dnn_handle_ownership_, Stream<gpu>::OwnHandle);

    TBlob data   = in_data[0];
    TBlob filter = in_data[1];
    TBlob out    = out_data[0];
    const TShape& dshape = data.shape_;
    const TShape& fshape = filter.shape_;
    const TShape& oshape = out.shape_;

    // allocate workspace
    if (!init_temp_size_) GetTempSize(ctx);
#if 0
    Tensor<gpu, 1, SrcType> workspace =
      ctx.requested[res_cnt++].get_space_typed<gpu, 1, SrcType>(mshadow::Shape1(workspace_), s);
#endif
    const int dev_id = ctx.run_ctx.ctx.dev_id;
    const int dev_mask = gpu::kDevMask;
    if (param_.layout == mshadow::kNCHW) {
#if 0
      TBlob data_(ctx.requested[res_cnt++].get_space_typed<gpu, 4, SrcType>(
          mshadow::Shape4(dshape[N], dshape[H], dshape[W], dshape[C]), s));
      TBlob filter_(ctx.requested[res_cnt++].get_space_typed<gpu, 4, SrcType>(
          mshadow::Shape4(fshape[N], fshape[H], fshape[W], fshape[C]), s));
#endif
      const size_t data_size = dshape.Size();
      const size_t weight_size = fshape.Size();
      const size_t output_size = oshape.Size();
      size_t total_temp_bytes = (workspace_ + data_size + weight_size) * sizeof(SrcType)
                              + output_size * (sizeof(DstType) + sizeof(int32_t));
      Tensor<gpu, 1, char> temp_space =
        ctx.requested[0].get_space_typed<gpu, 1, char>(mshadow::Shape1(total_temp_bytes), s);
      char* temp_dptr = temp_space.dptr_;
      TBlob data_(reinterpret_cast<SrcType*>(temp_dptr),
                  TShape({dshape[N], dshape[H], dshape[W], dshape[C]}),
                  dev_mask, DataType<SrcType>::kFlag, dev_id);
      temp_dptr += data_size * sizeof(SrcType);
      TBlob filter_(reinterpret_cast<SrcType*>(temp_dptr),
                    TShape({fshape[N], fshape[H], fshape[W], fshape[C]}),
                    dev_mask, DataType<SrcType>::kFlag, dev_id);
      temp_dptr += weight_size * sizeof(SrcType);
      
      // input:  [NCHW] => [NHWC](batch, in_height, in_width, in_channels)
      // filter: [NCHW] => [NHWC](out_channels, filter_height, filter_width, in_channels)
      TransposeImpl<gpu>(ctx.run_ctx, data,   data_,   TShape({N, H, W, C}));
      TransposeImpl<gpu>(ctx.run_ctx, filter, filter_, TShape({N, H, W, C}));
#if 0
      TBlob out_(ctx.requested[res_cnt++].get_space_typed<gpu, 4, DstType>(
          mshadow::Shape4(oshape[N], oshape[H], oshape[W], oshape[C]), s));
      TBlob out_tcast(ctx.requested[res_cnt++].get_space_typed<gpu, 4, int32_t>(
          mshadow::Shape4(oshape[N], oshape[H], oshape[W], oshape[C]), s));
#endif
      TBlob out_(reinterpret_cast<DstType*>(temp_dptr),
                 TShape({oshape[N], oshape[H], oshape[W], oshape[C]}),
                 dev_mask, DataType<DstType>::kFlag, dev_id);
      temp_dptr += output_size * sizeof(DstType);
      TBlob out_tcast(reinterpret_cast<int32_t*>(temp_dptr),
                      TShape({oshape[N], oshape[H], oshape[W], oshape[C]}),
                      dev_mask, DataType<int32_t>::kFlag, dev_id);
      temp_dptr += output_size * sizeof(int32_t);
      // input:  [NHWC](batch, in_height, in_width, in_channels)
      // filter: [HWNC](out_channels, filter_height, filter_width, in_channels)
      // output: [NHWC](batch, out_height, out_width, out_channels)

      CUDNN_CALL(hipdnnConvolutionForward(s->dnn_handle_,
                                         &alpha_,
                                         data_desc_,
                                         data_.dptr_,
                                         filter_desc_,
                                         filter_.dptr_,
                                         conv_desc_,
                                         algo_,
                                         temp_dptr,
                                         workspace_byte_,
                                         &beta_,
                                         out_desc_,
                                         out_.dptr_));

      Tensor<gpu, 1, DstType> out_tensor = out_.FlatTo1D<gpu, DstType>(s);
      Tensor<gpu, 1, int32_t> out_tcast_tensor = out_tcast.FlatTo1D<gpu, int32_t>(s);
      Assign(out_tcast_tensor, kWriteTo, mshadow::expr::tcast<int32_t>(out_tensor));
      // output: [NHWC](batch, out_height, out_width, out_channels) => [NCHW]
      TransposeImpl<gpu>(ctx.run_ctx, out_tcast, out, TShape({0, 3, 1, 2}));
    } else if (param_.layout == mshadow::kNHWC) {
      LOG(FATAL) << "Not implemented";
#if 0
      TBlob out_float(ctx.requested[res_cnt++].get_space_typed<gpu, 4, DstType>(
          mshadow::Shape4(oshape[N], oshape[H], oshape[W], oshape[C]), s));
      CUDNN_CALL(hipdnnConvolutionForward(s->dnn_handle_,
                                         &alpha_,
                                         data_desc_,
                                         data.dptr_,
                                         filter_desc_,
                                         filter.dptr_,
                                         conv_desc_,
                                         algo_,
                                         workspace.dptr_,
                                         workspace_byte_,
                                         &beta_,
                                         out_desc_,
                                         out_float.dptr_));
      Tensor<gpu, 1, DstType> out_float_tensor = out_float.FlatTo1D<gpu, DstType>(s);
      Tensor<gpu, 1, int32_t> out_tensor = out.FlatTo1D<gpu, int32_t>(s);
      Assign(out_tensor, kWriteTo, mshadow::expr::tcast<int32_t>(out_float_tensor));
#endif
    }

    // calculate the min/max range for out_data as it's a multiplication
    // of in_data[0] and in_data[1]. Need to rescale the min/max range of out_data
    // based on the min/max ranges of in_data[0] and in_data[1].
    const size_t num_inputs = param_.no_bias ? 2 : 3;
    mxnet_op::Kernel<QuantizationRangeForMultiplicationStruct, gpu>::Launch(s, 1,
      out_data[1].dptr<float>(), out_data[2].dptr<float>(),
       in_data[num_inputs].dptr<float>(),  in_data[num_inputs+1].dptr<float>(),
       in_data[num_inputs+2].dptr<float>(),  in_data[num_inputs+3].dptr<float>());

    if (!param_.no_bias) {
      CHECK_EQ(param_.layout, mshadow::kNCHW)
        << "quantized_conv2d only supports NCHW when there is a bias";
      const TBlob& bias = in_data[2];
      mxnet_op::Kernel<QuantizedBiasAddStruct, gpu>::Launch(s, out.Size(),
          bias.Size(), out.dptr<int32_t>(), bias.dptr<int8_t>(),
          out_data[1].dptr<float>(), out_data[2].dptr<float>(),
          in_data[7].dptr<float>(),  in_data[8].dptr<float>(),
          oshape[2] * oshape[3]);
    }
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_args) {
    LOG(FATAL) << "Not implemented";
  }


  void InitDescriptors(const Context& ctx,
                       const std::vector<TShape>& in_shape,
                       const std::vector<TShape>& out_shape) {
    TShape dshape =  in_shape[0];
    TShape kshape =  in_shape[1];
    TShape oshape = out_shape[0];
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&data_desc_));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc_));
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc_));

    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc_,
                                               param_.pad[0],
                                               param_.pad[1],
                                               param_.stride[0],
                                               param_.stride[1],
                                               1,
                                               1,
                                               HIPDNN_CROSS_CORRELATION,
                                               cmp_type_));

    CUDNN_CALL(hipdnnSetTensor4dDescriptor(data_desc_,
                                          format_,
                                          src_type_,
                                          dshape[N],
                                          dshape[C],
                                          dshape[H],
                                          dshape[W]));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc_,
                                          format_,
                                          dst_type_,
                                          oshape[N],
                                          oshape[C],
                                          oshape[H],
                                          oshape[W]));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc_,
                                          src_type_,
                                          format_,
                                          kshape[N],
                                          kshape[C],
                                          kshape[H],
                                          kshape[W]));
  }

  void GetTempSize(const OpContext& ctx) {
    CHECK(!init_temp_size_)
      << "GetTempSize should only be called once.";
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(s->dnn_handle_,
                                                       data_desc_,
                                                       filter_desc_,
                                                       conv_desc_,
                                                       out_desc_,
                                                       algo_,
                                                       &workspace_byte_));
    workspace_ = workspace_byte_ / sizeof(SrcType) + 1;
    init_temp_size_ = true;
  }


 private:
  bool init_temp_size_ = false;
  QuantizedConv2DParam param_;
  size_t workspace_;
  size_t workspace_byte_;
  hipdnnDataType_t src_type_;
  hipdnnDataType_t dst_type_;
  hipdnnDataType_t cmp_type_;
  hipdnnTensorFormat_t format_;
  hipdnnConvolutionDescriptor_t conv_desc_;
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnFilterDescriptor_t filter_desc_;
  hipdnnTensorDescriptor_t out_desc_;
  hipdnnConvolutionFwdAlgo_t algo_;
  uint32_t N, H, W, C;
  float alpha_ = 1.0f;
  float beta_ = 0.0f;
};  // class QuantizedReluCuDNNOp


template<>
Operator* CreateOp<gpu>(int dtype,
                        const Context& ctx,
                        const std::vector<TShape>& in_shape,
                        const std::vector<TShape>& out_shape,
                        const QuantizedConv2DParam& param) {
  Operator *op = NULL;
  op = new QuantizedConv2DCuDNNOp<int8_t, float, int32_t>(ctx,
    in_shape, out_shape, param);
  return op;
}

}  // namespace op
}  // namespace mxnet

